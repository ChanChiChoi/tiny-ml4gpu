#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "common/include/malloc_free.h"
#include "common/include/helper.cuh"
#include "ml/include/math/svd.h"


void
svd(float *A_device, const int Row_A, const int Col_A, const int lda,
    float *U_device, const int Row_U, const int Col_U,
    float *S_device, const int Length,
    float *VT_device, const int Row_VT, const int Col_VT){

  assert(Row_A >= Col_A);
  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  float *rwork_device = NULL;

  int lwork = 0;
  int info_gpu = 0;

  CHECK_CALL_DEFAULT(hipsolverDnCreate(&cusolverH));
  CHECK_CALL_DEFAULT(hipblasCreate(&cublasH));

  int *devInfo_device;
  devInfo_device = DEVICE_MALLOC(devInfo_device,sizeof(int));

  //hipsolverDnSgesvd_bufferSize single precision
  CHECK_CALL_DEFAULT(hipsolverDnSgesvd_bufferSize(cusolverH,Row_A,Col_A,&lwork));

  float *work_device;
  work_device = DEVICE_MALLOC(work_device,sizeof(float)*lwork);

  //step 4:compute svd
  signed char jobu = 'A';
  signed char jobvt = 'A';
  //hipsolverDnSgesvd single precision
  CHECK_CALL_DEFAULT(hipsolverDnSgesvd(cusolverH, jobu, jobvt, 
                          Row_A, Col_A,
                          A_device, lda,
                          S_device, U_device,lda, 
                          VT_device, lda, work_device, lwork, rwork_device, devInfo_device));

  CHECK_CALL_DEFAULT(hipDeviceSynchronize());
 
  DEVICE_TO_HOST_FREE(&info_gpu, devInfo_device, sizeof(int));  

  DEVICE_FREE(work_device);
  DEVICE_FREE(rwork_device);

  hipblasDestroy(cublasH);
  hipsolverDnDestroy(cusolverH);
    
  return ;
}

void
svd(double *A_device, const int Row_A, const int Col_A, const int lda,
    double *U_device, const int Row_U, const int Col_U,
    double *S_device, const int Length,
    double *VT_device, const int Row_VT, const int Col_VT){

  assert(Row_A >= Col_A);
  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  double *rwork_device = NULL;

  int lwork = 0;
  int info_gpu = 0;

  CHECK_CALL_DEFAULT(hipsolverDnCreate(&cusolverH));
  CHECK_CALL_DEFAULT(hipblasCreate(&cublasH));

  int *devInfo_device;
  devInfo_device = DEVICE_MALLOC(devInfo_device,sizeof(int));

  //hipsolverDnDgesvd_bufferSize double precision
  CHECK_CALL_DEFAULT(hipsolverDnDgesvd_bufferSize(cusolverH,Row_A,Col_A,&lwork));

  double *work_device;
  work_device = DEVICE_MALLOC(work_device,sizeof(double)*lwork);

  //step 4:compute svd
  signed char jobu = 'A';
  signed char jobvt = 'A';
  //hipsolverDnDgesvd double precision
  CHECK_CALL_DEFAULT(hipsolverDnDgesvd(cusolverH, jobu, jobvt, 
                          Row_A, Col_A,
                          A_device, lda,
                          S_device, U_device,lda, 
                          VT_device, lda, work_device, lwork, rwork_device, devInfo_device));

  CHECK_CALL_DEFAULT(hipDeviceSynchronize());
 
  DEVICE_TO_HOST_FREE(&info_gpu, devInfo_device, sizeof(int));  

  DEVICE_FREE(work_device);
  DEVICE_FREE(rwork_device);

  hipblasDestroy(cublasH);
  hipsolverDnDestroy(cusolverH);
    
  return ;
}
