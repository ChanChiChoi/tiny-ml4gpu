#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <math_functions.hpp>
#include "common/include/type.h"
#include "common/include/common.h"
//#include "common/include/malloc_free.h"
#include "ml/include/math/matrix.h"

# define TILE_HEIGHT 32
# define TILE_WIDTH 32

//template ===
//matrix_mul

template<typename T> __device__ T
scalar_multiply(T x, T y){
  return x*y;
}

template<typename T> __device__ T
scalar_mse(T x, T y){
    T tmp = abs(x-y);
    return tmp*tmp;
}

template<typename T> __device__ T
scalar_operation(T x, T y, const int op){
  /*
   this function used to be entrance of how to handle two scalar
   1 - x * y
   2 - |x-y|^2

  */
  T ans = T(0);
  if (op == 1){
      ans = scalar_multiply<T>(x,y);
  }else if(op == 2){
      ans = scalar_mse(x,y); 
  }
  return ans;
}


template<typename T> __global__ void
matrix_mul(T * Md, u32 Row_Md, u32 Col_Md,
           T * Nd, u32 Row_Nd, u32 Col_Nd,
           T * Pd, u32 Row_Pd, u32 Col_Pd,
           const int op = 1
           ){
    
    /*
     each thread has two task:
    1 - fetch data into shared mem;
    2 - calc the data by steps , then get result of Pd

*/
    // row = height = x;   
    // col = width = y
    assert(Col_Md == Row_Nd);

    __shared__ T Mds[TILE_HEIGHT][TILE_WIDTH];
    __shared__ T Nds[TILE_HEIGHT][TILE_WIDTH];

    u32 bx = blockIdx.x;
    u32 by = blockIdx.y;
    u32 tx = threadIdx.x;
    u32 ty = threadIdx.y;

    // split Md by TILE_WIDTH, so each blocksize equal TILE_WIDTH
    // current we create the Row,Col in Pd
    u32 Row = by*TILE_HEIGHT + ty;
    u32 Col = bx*TILE_WIDTH + tx;

    // here should not use  "(Row < Row_Pd && Col < Col_Pd)", because we need other thread
    // to fetch data into shared mem.

    T Pvalue = 0;

    
    // for cur tx,ty only care Col of Md and Row of Nd
    for(u32 m = 0; m < ceil((double)Col_Md/TILE_WIDTH); ++m){
        // get the data again and again
        // if cur tx,ty is exceend of Md,Nd, then it should be exit early,
        // so it will not run here
        const u32 ind_bef_Md = Row*Col_Md;
        const u32 ind_x_Md = m*TILE_WIDTH + tx;

        const u32 ind_y_Nd = m*TILE_HEIGHT + ty;

        // if cur x is exceed col of md, then skip
        if (ind_x_Md < Col_Md)
            Mds[ty][tx] = Md[ind_bef_Md + ind_x_Md];
 
        // if cur y is exceed row of nd, then skip
        if (ind_y_Nd  < Row_Nd)
            Nds[ty][tx] = Nd[ind_y_Nd*Col_Nd + Col];

        // if cur thread can do nothing, then exit
        if (ind_x_Md >= Col_Md && ind_y_Nd >= Row_Nd)
            return ;

        __syncthreads();

       // if cur thread' task contain create pd result, it need handle follow code
       if (Row < Row_Pd && Col < Col_Pd){ 

           u32 ind_max_TILE;
           if ((m+1)*TILE_WIDTH <= Col_Md)
               ind_max_TILE = TILE_WIDTH;
           else
               ind_max_TILE = Col_Md - m*TILE_WIDTH;
           // calc the point
           for(u32 k = 0; k < ind_max_TILE; ++k){
              //Pvalue += Mds[ty][k] * Nds[k][tx];
              Pvalue += scalar_operation(Mds[ty][k], Nds[k][tx],op);
           }
       }

       __syncthreads();

   } 

   if(Row >= Row_Pd || Col >= Col_Pd)
        return ;
   // put the result into origin location of Pd
   Pd[Row*Col_Pd + Col] = Pvalue;
}

template<typename T> void
matrix_mul_launch(T * Md, u32 Row_Md, u32 Col_Md,
           T * Nd, u32 Row_Nd, u32 Col_Nd,
           T * Pd, u32 Row_Pd, u32 Col_Pd,
           const int op = 1){

    dim3 grid(MAX(1, (size_t)ceil((double)Col_Pd/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row_Pd/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);

    matrix_mul<T><<<grid, block>>>(Md, Row_Md, Col_Md,
           Nd, Row_Nd, Col_Nd,
           Pd, Row_Pd, Col_Pd,
           op);
}


template<typename T> __global__ void
matrix_transpose(T * mat_src, u32 Row_src, u32 Col_src,
                 T * mat_dst, u32 Row_dst, u32 Col_dst){

    assert(Row_src*Col_src == Row_dst*Col_dst);
    u32 idy = blockIdx.y*blockDim.y + threadIdx.y;
    u32 idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idy >= Row_src || idx >= Col_src)
        return ;

    mat_dst[idx*Col_dst + idy] = mat_src[idy*Col_src + idx];

}


template<typename T> void
matrix_transpose_launch(T *mat_src, u32 Row_src, u32 Col_src,
                 T * mat_dst, u32 Row_dst, u32 Col_dst){

    dim3 grid(MAX(1, (size_t)ceil((double)Col_src/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row_src/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);

    matrix_transpose<T><<<grid, block>>>(mat_src, Row_src, Col_src,
                                   mat_dst, Row_dst, Col_dst);
}

template<typename T>__global__ void
matrix_divide_scalar(T *mat, u32 Row, u32 Col, u32 scalar){

    u32 idy = blockIdx.y*blockDim.y + threadIdx.y;
    u32 idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idy >= Row || idx >= Col)
        return ;

    mat[idy*Col+idx] /= scalar;
}

template<typename T> void
matrix_divide_scalar_launch(T *mat, u32 Row, u32 Col, u32 scalar){

    dim3 grid(MAX(1, (size_t)ceil((double)Col/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);
  
    matrix_divide_scalar<T><<<grid, block>>>(mat, Row, Col, scalar);

}

template<typename T> __global__ void
matrix_subblock(T *big, u32 Row_big, u32 Col_big,
                T *small, u32 Row_sm, u32 Col_sm,
                u32 rmin, u32 cmin, u32 rmax, u32 cmax){
    // rmin base on 0.
    u32 idy = blockIdx.y*blockDim.y + threadIdx.y;
    u32 idx = blockIdx.x*blockDim.x + threadIdx.x;
 
    assert(rmax - rmin == Row_sm);
    assert(cmax - cmin == Col_sm);

    if(idy >= Row_sm || idx >= Col_sm)
        return ;

    small[idy*Col_sm+idx] = big[(rmin+idy)*Col_big+cmin+idx];
}

template<typename T> void
matrix_subblock_launch(T *big, u32 Row_big, u32 Col_big,
                       T *small, u32 Row_sm, u32 Col_sm,
                       u32 rmin, u32 cmin, u32 rmax, u32 cmax){

    dim3 grid(MAX(1, (size_t)ceil((double)Col_sm/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row_sm/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);

    matrix_subblock<T><<<grid, block>>>(big, Row_big, Col_big,
                                       small, Row_sm, Col_sm,
                                      rmin, cmin, rmax, cmax);

}
// ==========cov


//================ 
void
matrix_mul_cpu(float *Md, u32 Row_Md, u32 Col_Md,
               float *Nd, u32 Row_Nd, u32 Col_Nd,
               float *Pd, u32 Row_Pd, u32 Col_Pd,
               const int op = 1){

    matrix_mul_launch<float>(Md, Row_Md, Col_Md,
               Nd, Row_Nd, Col_Nd,
               Pd, Row_Pd, Col_Pd,
               op);

}

void
matrix_transpose_cpu(float *mat_src, u32 Row_src, u32 Col_src,
                     float * mat_dst, u32 Row_dst, u32 Col_dst){

    matrix_transpose_launch<float>(mat_src, Row_src, Col_src,
                      mat_dst, Row_dst, Col_dst);
}

void
matrix_divide_scalar_cpu(float *mat, u32 Row, u32 Col, u32 scalar){

    matrix_divide_scalar_launch<float>(mat, Row, Col, scalar);
}

void
matrix_subblock_cpu(float *big, u32 Row_big, u32 Col_big,
                float *small, u32 Row_sm, u32 Col_sm,
                u32 rmin, u32 cmin, u32 rmax, u32 cmax){

    matrix_subblock_launch<float>(big, Row_big, Col_big,
                           small, Row_sm, Col_sm,
                           rmin, cmin, rmax, cmax);
}

//void
//cov_cpu(float *mat, u32 Row_mat, u32 Col_mat,
//        float *mat_cov, u32 Row_mat_cov, u32 Col_mat_cov){
//    
//    //1 - malloc one matrix
//    size_t size = sizeof(float)*Row_mat*Col_mat;
//    float *mat_T_device = NULL;
//    mat_T_device = DEVICE_MALLOC(mat_T_device, size);
//
//    //2 - transpose
//    u32 Row_mat_T = Col_mat;
//    u32 Col_mat_T = Row_mat;
//    matrix_transpose_cpu(mat,Row_mat, Col_mat,
//                  mat_T_device, Row_mat_T, Col_mat_T);
//
//    //3 - matrix_mul
//
//    matrix_mul_cpu(mat_T_device,Row_mat_T, Col_mat_T,
//                   mat, Row_mat, Col_mat,
//                   mat_cov, Row_mat_cov, Col_mat_cov,1);
//
//    DEVICE_FREE(mat_T_device);
//
//    //4 - divide (n-1) samples;
//    size_t n_1 = MAX(1,Row_mat-1);
//    matrix_divide_scalar_cpu(mat_cov, Row_mat_cov, Col_mat_cov, n_1);
//
//}
