#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <math_functions.hpp>
#include "common/include/type.h"
#include "common/include/common.h"
//#include "common/include/malloc_free.h"
#include "ml/include/math/matrix.h"

# define TILE_HEIGHT 32
# define TILE_WIDTH 32

/*
template === one scalar operation
*/
template<typename T> __device__ T
scalar_sqrt(T x){
    return sqrt(x);
}


template<typename T> __device__ T
scalar_operation1(T x, const int op){
  /*
   this function used to be entrance of how to handle one scalar
   1 - sqrt(x)

  */
  T ans = T(0);
  if (op == 1){
      ans = scalar_sqrt<T>(x);
  }
  return ans;
}

/*
template === two scalar operation
*/
template<typename T> __device__ T
scalar_multiply(T x, T y){
  return x*y;
}

template<typename T> __device__ T
scalar_mse(T x, T y){
    T tmp = abs(x-y);
    return tmp*tmp;
}

template<typename T> __device__ T
scalar_divide(T x, T y){
  return x-y;
}

template<typename T> __device__ T
scalar_gaussian(T x, T sigma){
  // T should not be int data type, in case of return 0
  return exp(-x*x/(2*sigma*sigma));
   
}


template<typename T> __device__ T
scalar_operation2(T x, T y, const int op){
  /*
   this function used to be entrance of how to handle two scalar
   1 - x * y
   2 - |x-y|^2

  */
  T ans = T(0);
  if (op == 1){
      ans = scalar_multiply<T>(x,y);
  }else if(op == 2){
      ans = scalar_mse<T>(x,y); 
  }else if(op == 3){
      ans = scalar_divide<T>(x,y);
  }else if(op == 4){
      ans = scalar_gaussian<T>(x,y);
  }
  return ans;
}


//=============
template<typename T> __global__ void
matrix_mul(T * Md, u32 Row_Md, u32 Col_Md,
           T * Nd, u32 Row_Nd, u32 Col_Nd,
           T * Pd, u32 Row_Pd, u32 Col_Pd,
           const int op = 1
           ){
    
    /*
     each thread has two task:
    1 - fetch data into shared mem;
    2 - calc the data by steps , then get result of Pd

*/
    // row = height = x;   
    // col = width = y
    assert(Col_Md == Row_Nd);

    __shared__ T Mds[TILE_HEIGHT][TILE_WIDTH];
    __shared__ T Nds[TILE_HEIGHT][TILE_WIDTH];

    u32 bx = blockIdx.x;
    u32 by = blockIdx.y;
    u32 tx = threadIdx.x;
    u32 ty = threadIdx.y;

    // split Md by TILE_WIDTH, so each blocksize equal TILE_WIDTH
    // current we create the Row,Col in Pd
    u32 Row = by*TILE_HEIGHT + ty;
    u32 Col = bx*TILE_WIDTH + tx;

    // here should not use  "(Row < Row_Pd && Col < Col_Pd)", because we need other thread
    // to fetch data into shared mem.

    T Pvalue = 0;

    
    // for cur tx,ty only care Col of Md and Row of Nd
    for(u32 m = 0; m < ceil((double)Col_Md/TILE_WIDTH); ++m){
        // get the data again and again
        // if cur tx,ty is exceend of Md,Nd, then it should be exit early,
        // so it will not run here
        const u32 ind_bef_Md = Row*Col_Md;
        const u32 ind_x_Md = m*TILE_WIDTH + tx;

        const u32 ind_y_Nd = m*TILE_HEIGHT + ty;

        // if cur x is exceed col of md, then skip
        if (ind_x_Md < Col_Md)
            Mds[ty][tx] = Md[ind_bef_Md + ind_x_Md];
 
        // if cur y is exceed row of nd, then skip
        if (ind_y_Nd  < Row_Nd)
            Nds[ty][tx] = Nd[ind_y_Nd*Col_Nd + Col];

        // if cur thread can do nothing, then exit
        if (ind_x_Md >= Col_Md && ind_y_Nd >= Row_Nd)
            return ;

        __syncthreads();

       // if cur thread' task contain create pd result, it need handle follow code
       if (Row < Row_Pd && Col < Col_Pd){ 

           u32 ind_max_TILE;
           if ((m+1)*TILE_WIDTH <= Col_Md)
               ind_max_TILE = TILE_WIDTH;
           else
               ind_max_TILE = Col_Md - m*TILE_WIDTH;
           // calc the point
           for(u32 k = 0; k < ind_max_TILE; ++k){
              //Pvalue += Mds[ty][k] * Nds[k][tx];
              Pvalue += scalar_operation2(Mds[ty][k], Nds[k][tx],op);
           }
       }

       __syncthreads();

   } 

   if(Row >= Row_Pd || Col >= Col_Pd)
        return ;
   // put the result into origin location of Pd
   Pd[Row*Col_Pd + Col] = Pvalue;
}


template<typename T> void
matrix_mul_launch(T * Md, u32 Row_Md, u32 Col_Md,
           T * Nd, u32 Row_Nd, u32 Col_Nd,
           T * Pd, u32 Row_Pd, u32 Col_Pd,
           const int op = 1){

    dim3 grid(MAX(1, (size_t)ceil((double)Col_Pd/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row_Pd/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);

    matrix_mul<T><<<grid, block>>>(Md, Row_Md, Col_Md,
           Nd, Row_Nd, Col_Nd,
           Pd, Row_Pd, Col_Pd,
           op);
}

/*
function: matrix_transpose

*/
template<typename T> __global__ void
matrix_transpose(T * mat_src, u32 Row_src, u32 Col_src,
                 T * mat_dst, u32 Row_dst, u32 Col_dst){

    assert(Row_src*Col_src == Row_dst*Col_dst);
    u32 idy = blockIdx.y*blockDim.y + threadIdx.y;
    u32 idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idy >= Row_src || idx >= Col_src)
        return ;

    mat_dst[idx*Col_dst + idy] = mat_src[idy*Col_src + idx];

}


template<typename T> void
matrix_transpose_launch(T *mat_src, u32 Row_src, u32 Col_src,
                 T * mat_dst, u32 Row_dst, u32 Col_dst){

    dim3 grid(MAX(1, (size_t)ceil((double)Col_src/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row_src/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);

    matrix_transpose<T><<<grid, block>>>(mat_src, Row_src, Col_src,
                                   mat_dst, Row_dst, Col_dst);
}


/*
function: matrix_scalar_self
*/
template<typename T> __global__ void
matrix_scalar_self(T *mat, u32 Row, u32 Col, const int op){

    u32 idy = blockIdx.y*blockDim.y + threadIdx.y;
    u32 idx = blockIdx.x*blockDim.x + threadIdx.x;

    
    if (idy >= Row || idx >= Col)
        return ;
    T x = mat[idy*Col+idx];
    mat[idy*Col+idx] = scalar_operation1(x,op);
}



template<typename T> void
matrix_scalar_self_launch(T *mat, u32 Row, u32 Col,const int op=1){

    dim3 grid(MAX(1, (size_t)ceil((double)Col/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);
  
    matrix_scalar_self<T><<<grid, block>>>(mat, Row, Col,op);

}


/*
function: matrix_scalar
*/
template<typename T>__global__ void
matrix_scalar(T *mat, u32 Row, u32 Col, u32 scalar, const int op){

    u32 idy = blockIdx.y*blockDim.y + threadIdx.y;
    u32 idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idy >= Row || idx >= Col)
        return ;
    T x = mat[idy*Col+idx];
    mat[idy*Col+idx] = scalar_operation2(x,T(scalar),op);
//    mat[idy*Col+idx] /= scalar;
}

template<typename T> void
matrix_scalar_launch(T *mat, u32 Row, u32 Col, u32 scalar,const int op=3){

    dim3 grid(MAX(1, (size_t)ceil((double)Col/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);
  
    matrix_scalar<T><<<grid, block>>>(mat, Row, Col, scalar,op);

}


/*
function: matrix_subblock
*/
template<typename T> __global__ void
matrix_subblock(T *big, u32 Row_big, u32 Col_big,
                T *small, u32 Row_sm, u32 Col_sm,
                u32 rmin, u32 cmin, u32 rmax, u32 cmax){
    // rmin base on 0.
    u32 idy = blockIdx.y*blockDim.y + threadIdx.y;
    u32 idx = blockIdx.x*blockDim.x + threadIdx.x;
 
    assert(rmax - rmin == Row_sm);
    assert(cmax - cmin == Col_sm);

    if(idy >= Row_sm || idx >= Col_sm)
        return ;

    small[idy*Col_sm+idx] = big[(rmin+idy)*Col_big+cmin+idx];
}

template<typename T> void
matrix_subblock_launch(T *big, u32 Row_big, u32 Col_big,
                       T *small, u32 Row_sm, u32 Col_sm,
                       u32 rmin, u32 cmin, u32 rmax, u32 cmax){

    dim3 grid(MAX(1, (size_t)ceil((double)Col_sm/TILE_HEIGHT)),
              MAX(1, (size_t)ceil((double)Row_sm/TILE_WIDTH)) );
    dim3 block(TILE_WIDTH, TILE_HEIGHT);

    matrix_subblock<T><<<grid, block>>>(big, Row_big, Col_big,
                                       small, Row_sm, Col_sm,
                                      rmin, cmin, rmax, cmax);

}



//================ 
/*
function: matrix_mul_cpu
*/
void
matrix_mul_cpu(float *Md, u32 Row_Md, u32 Col_Md,
               float *Nd, u32 Row_Nd, u32 Col_Nd,
               float *Pd, u32 Row_Pd, u32 Col_Pd,
               const int op){

    matrix_mul_launch<float>(Md, Row_Md, Col_Md,
               Nd, Row_Nd, Col_Nd,
               Pd, Row_Pd, Col_Pd,
               op);

}

/*
function: matrix_transpose_cpu
*/
void
matrix_transpose_cpu(float *mat_src, u32 Row_src, u32 Col_src,
                     float * mat_dst, u32 Row_dst, u32 Col_dst){

    matrix_transpose_launch<float>(mat_src, Row_src, Col_src,
                      mat_dst, Row_dst, Col_dst);
}

/*
function: matrix_divide_scalar_cpu
*/
void
matrix_divide_scalar_cpu(float *mat, u32 Row, u32 Col, u32 scalar){
    matrix_scalar_launch<float>(mat, Row, Col, scalar,3);
}

/*
function: matrix_gaussian_scalar_cpu
*/
void
matrix_gaussian_scalar_cpu(float *mat, u32 Row, u32 Col, u32 scalar_sigma){
    matrix_scalar_launch<float>(mat, Row, Col, scalar_sigma,4);
}


/*
function: matrix_subblock_cpu
*/
void
matrix_subblock_cpu(float *big, u32 Row_big, u32 Col_big,
                float *small, u32 Row_sm, u32 Col_sm,
                u32 rmin, u32 cmin, u32 rmax, u32 cmax){

    matrix_subblock_launch<float>(big, Row_big, Col_big,
                           small, Row_sm, Col_sm,
                           rmin, cmin, rmax, cmax);
}


/*
function: matrix_sqrt_cpu
*/
void
matrix_scalar_sqrt_cpu(float *mat, u32 Row_mat, u32 Col_mat){

    matrix_scalar_self_launch<float>(mat, Row_mat, Col_mat, 1);
}

