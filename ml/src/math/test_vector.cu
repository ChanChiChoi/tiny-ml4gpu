#include "vector.cu"

int
main(){
     
    size_t rows = 10000;
    size_t cols = 10000;
    size_t size = sizeof(float)*rows*cols;
    float *mat = (float *)malloc(size);

    float *md_device;
    hipMalloc((void **)&md_device,size);
    hipMemcpy(md_device, mat, size, hipMemcpyHostToDevice);


    size_t cols_vec = cols;
    size_t size1 = sizeof(float)*cols_vec;
    float *vec = (float *)malloc(size1);

    float *vec_device;
    hipMalloc((void **)&vec_device,size1);
    hipMemcpy(vec_device, vec, size1, hipMemcpyHostToDevice);

    vector_repeat_by_rows_cpu(md_device, rows, cols,
                vec_device, cols_vec);
    hipFree(md_device);
    hipFree(vec_device);
    free(mat);
    free(vec);
    return 0;
}
