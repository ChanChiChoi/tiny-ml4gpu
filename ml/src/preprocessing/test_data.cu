#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include "data.cu"
using namespace std::chrono;

int
main(){

    size_t size = 200*sizeof(float);
    float *mat = (float *)malloc(size);
    for(u32 i=0;i<200;i++)
        mat[i] = i;

//    for(u32 i=0; i<20;i++){
//        printf("val %d %f\n",i,mat[i]);
//    }

    u32 a = 16;
    u32 b = 50;
    if(a<=b)
     printf("hello\n==============");
    float *mat_device = NULL;
    hipMalloc((void **)mat_device,size);
    hipMemcpy(mat_device, mat, size, hipMemcpyHostToDevice);

    size_t size1 = 50*sizeof(float);
    float *mean = (float *)malloc(size1);
    float *mean_device = nullptr;
    hipMalloc((void **)mean_device,size1);
    hipMemcpy(mean_device, mean, size1, hipMemcpyHostToDevice);
    

    auto t0 = high_resolution_clock::now();
    mean_by_rows_cpu(mat_device,mean_device, 4,50);
    hipDeviceSynchronize();
    auto t1 = high_resolution_clock::now();

    hipMemcpy(mat, mat_device, size, hipMemcpyDeviceToHost);
    hipMemcpy(mean, mean_device, size1, hipMemcpyDeviceToHost);

//
    for(u32 i=30; i<40;i++){
        printf("val %d %f %f\n",i,mat[i], mean[i]);
    }

    hipFree(mean_device);
    hipFree(mat_device);
    printf("take time %d\n",duration_cast<milliseconds>(t1-t0).count());
    free(mat);
    free(mean);
}

