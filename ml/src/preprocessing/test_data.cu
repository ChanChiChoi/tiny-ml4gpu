#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <chrono>

#include "data.cu"
#include "common/include/helper.cuh"
//using namespace std::chrono;

int
main(){

    size_t rows = 4;
    size_t cols = 50;
    size_t size = rows*cols*sizeof(float);
    float *mat = (float *)malloc(size);
    for(u32 i=0;i<cols;i++)
        mat[i] = i;

//    for(u32 i=0; i<20;i++){
//        printf("val %d %f\n",i,mat[i]);
//    }

    float *mat_device;
    hipMalloc((void **)&mat_device,size);
    hipMemcpy(mat_device, mat, size, hipMemcpyHostToDevice);

    size_t size1 = cols*sizeof(float);
    float *mean = (float *)malloc(size1);
    float *mean_device = nullptr;
    hipMalloc((void **)&mean_device,size1);
    hipMemcpy(mean_device, mean, size1, hipMemcpyHostToDevice);
    

    float *std = (float *)malloc(size1);
    float *std_device = nullptr;
    hipMalloc((void **)&std_device, size1);
    hipMemcpy(std_device, std, size1, hipMemcpyHostToDevice);
  

  //  auto t0 = high_resolution_clock::now();
    mean_by_rows_cpu(mat_device,mean_device, rows,cols);
    normalization_by_rows_cpu(mat_device,mean_device,std_device, rows,cols);
    hipDeviceSynchronize();
  //  auto t1 = high_resolution_clock::now();

    hipMemcpy(mat, mat_device, size, hipMemcpyDeviceToHost);
    hipMemcpy(mean, mean_device, size1, hipMemcpyDeviceToHost);
    hipMemcpy(std, std_device, size1, hipMemcpyDeviceToHost);

//
    for(u32 i=0; i<20;i++){
        printf("val %d %f %f %f\n",i,mat[i], mean[i], std[i]);
    }

    hipFree(mean_device);
    hipFree(mat_device);
    hipFree(std_device);
   // printf("take time %d\n",duration_cast<milliseconds>(t1-t0).count());
    free(mat);
    free(mean);
    free(std);
    return 0;
}

