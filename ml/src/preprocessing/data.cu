#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math_functions.hpp>
#include "common/include/type.h"
#include "common/include/common.h"

//====================template
// calc the mean by row dimension
template<typename T> __global__ void
mean_by_rows(T *mat_device, T *mean_vec, u32 rows, u32 cols){

    u32 idy = blockIdx.y*gridDim.y + threadIdx.y;
    u32 idx = blockIdx.x*gridDim.x + threadIdx.x;

    u32 thread_idx = idy*(gridDim.x*blockIdx.x) + idx;

    if(thread_idx < cols){
        T mean = (T)0;
        T cur_val = (T)0;
        for (u32 i = 0; i < rows; i++){
            cur_val = mat_device[i*cols+thread_idx];
            // in case of sum is too big
            mean = mean*((float)i/(i+1)) +  cur_val/(double)(i+1);
        }

        mean_vec[thread_idx] = mean;
    }
}

// each row subtract the mean vector
template<typename T> __global__ void
zero_mean_by_rows(T *mat_device, T *mean_vec, u32 rows, u32 cols){

    u32 idy = blockIdx.y*gridDim.y + threadIdx.y;
    u32 idx = blockIdx.x*gridDim.x + threadIdx.x;


    if(idx < cols && idy < rows){
        u32 val_idx = idy*cols + idx;
        mat_device[val_idx] -= mean_vec[idx];
    }

}

// calc the std by rows dimension
template<typename T> __global__ void
std_by_rows(T *mat_device, T *mean_vec, T *std_vec, u32 rows, u32 cols){

    u32 idy = blockIdx.y*gridDim.y + threadIdx.y;
    u32 idx = blockIdx.x*gridDim.x + threadIdx.x;

    u32 thread_idx = idy*(gridDim.x*blockIdx.x) + idx;
    if (thread_idx >= cols)
        return ;
    
    T cur_mean = mean_vec[thread_idx];
    T cur_val;
    T dif;
    T std;
    for (size_t i = 0; i< rows; i++){
        cur_val = mat_device[i*cols+thread_idx];
        dif = abs(cur_val - cur_mean);
        // in case of std sum is bigger than limits
        std = std*((float)i/(i+1)) +  dif*dif/(double)(i+1);
    }
    std_vec[thread_idx] = sqrt(std);

} 


// each row divide the std vector
template<typename T> __global__ void
one_std_by_rows(T *mat_device, T *std_vec, u32 rows, u32 cols){

    u32 idy = blockIdx.y*gridDim.y + threadIdx.y;
    u32 idx = blockIdx.x*gridDim.x + threadIdx.x;


    if(idx < cols && idy < rows){
        u32 val_idx = idy*cols + idx;
        mat_device[val_idx] /= std_vec[idx];
    }

}

//===========launch
//export the function for be called by host
void
mean_by_rows_launch(float *mat_device, float *mean_device, u32 rows, u32 cols){


    const u32 COLS = 256;
    dim3 grid0( MAX(1,ceil((double)cols/COLS)) );
    dim3 block0(COLS);

    mean_by_rows<float><<<grid0, block0>>>(mat_device, mean_device, rows, cols);

    const u32 block_size = 32;
    dim3 block1(block_size, block_size);

    dim3 grid1(MAX(1, ceil((double)cols/block_size)),
              MAX(1, ceil((double)rows/block_size)));

    zero_mean_by_rows<float><<<grid1,block1>>>(mat_device, mean_device, rows, cols);
}

void
normalization_by_rows_launch(float *mat_device, float *mean_device, float *std_device, u32 rows, u32 cols){

    mean_by_rows_launch(mat_device, mean_device, rows, cols);
    
    const u32 COLS = 256;
    dim3 grid0( MAX(1, ceil(double)cols/COLS));
    dim3 block0(COLS);

    std_by_rows<float><<<grid0, block0>>>(mat_device, mean_vec, std_device, rows, cols);

    const u32 block_size = 32;
    dim3 block1(block_size, block_size);
    dim3 grid1(MAX(1, ceil(double(cols)/block_size)),
               MAX(1, ceil(double(rows)/blocks_size)));

    one_std_by_rows<float><<<grid1, block1>>>(mat_device, std_device, rows, cols);
    

}
//===============export to host
void
mean_by_rows_cpu(float *mat_device, float *mean_device, u32 rows, u32 cols){

    mean_by_rows_launch(mat_device, mean_device, rows, cols);
}

void
normalization_by_rows_cpu(float *mat_device, float *mean_device, float *std_device, u32 rows, u32 cols){
    normalization_by_rows_launch(mat_device, mean_device, std_device, rows, cols);
}
/*
int
main(){

    size_t size = 200*sizeof(float);
    float *mat = (float *)malloc(size);
    for(u32 i=0;i<200;i++)
        mat[i] = i;


    float *mat_device = host_to_device_malloc(mat,size);

    size_t size1 = 50*sizeof(float);
    float *mean = (float *)malloc(size1);
    float *mean_device = host_to_device_malloc(mean,size1);

    auto t0 = high_resolution_clock::now();
    mean_by_rows_cpu(mat_device,mean_device, 4,50);
    hipDeviceSynchronize();
    auto t1 = high_resolution_clock::now();
    device_to_host_free(mat,mat_device,size);
    device_to_host_free(mean,mean_device,size1);
    printf("take time %d\n",duration_cast<milliseconds>(t1-t0).count());

//
    for(u32 i=0; i<20;i++){
        printf("val %d %f\n",i,mat[i]);
    }

}
*/
