#include "hip/hip_runtime.h"
/*
the file imitates sklearn.preprocessing.data
*/
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<limits>
#include<assert.h>
#include<cmath>

#include "ML/preprocessing/data.h"

#include "common/helper.h"
#include "common/buffer_info.h"
#include "common/malloc_free.h"

#define MAX(x,y) ((x)>(y) ? (x): (y))
#define MIN(x,y) ((x)>(y) ? (y): (x))


/* get the [n by m] matrix's maxValue vector and minValue vector by col dimension,
means maxVal vector is [1 by m], minVal vector is [1 by m]*/
template<class T> __global__ void
_get_minmax(T *mat, T *min, T *max, unsigned int cols, unsigned int rows, T min_val, T max_val){

    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int thread_idx = (gridDim.x * blockDim.x)*idy +idx;

    // launch col numbers cuda threads to handle the matrix.
    // each cuda thread handle one col dimension
    if (thread_idx < cols){
        T min_l = max_val;
        T max_l = min_val;
    
        T tmp = (T)0;
        for (unsigned int i = 0; i < rows; i++){
            tmp = *(mat + cols*i + thread_idx);
            min_l = MIN(tmp, min_l);
            max_l = MAX(tmp, max_l);
        } 
        min[thread_idx] = min_l;
        max[thread_idx] = max_l;
    }
}


template<typename T> vector<T *>
get_minmax(Buf &buf){
    
    assert(buf.ptr_device != NULL);

    auto cols = buf.rows_cols()[1];
    size_t size_min = cols * buf.itemsize;
    size_t size_max = size_min;
    
    float *min_d = device_malloc<float>(size_min);
    float *max_d = device_malloc<float>(size_max);

    float min_val = std::numeric_limits<float>::min();
    float max_val = std::numeric_limits<float>::max();

    _get_minmax<T><<<grid_size, block_size>>>((float *)buf.ptr_device, min_d, max_d, cols, rows, min_val, max_val);

    


}


/*normaliza the [n by m] matrix, use col cuda threads*/
template<class T> __global__ void
minmax_scale_cuda(T *mat, T *min, T *max, unsigned int cols, unsigned int rows, T feature_min, T feature_max){
/* feature_range should bigger than 0*/

    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int thread_idx = (gridDim.x * blockDim.x)*idy +idx;

    if (thread_idx < cols){
       T min_l = min[thread_idx];
       T max_l = max[thread_idx];

       T range = max_l - min_l;
       
       T feature_range_l = feature_max - feature_min; // should not be 0, if it's 0, then replace with 1
       assert(feature_range_l > 0 );
       float scale = feature_range_l / range; 
       float min_  = feature_min - min_l * scale;
       
       T tmp = (T)0;
       for(unsigned int i = 0; i<rows; i++){
           tmp = *(mat + cols*i + thread_idx);
           tmp *= scale;
           tmp += min_;
           *(mat + cols*i + thread_idx) = (T)tmp;

       }

    }

}


//TODO: need extra min_d  and max_d.
//TODO: create minmax_scale class

template<class T> void
_minmax_scale_cpu(Buf &buf, unsigned int rows, unsigned int cols, T feature_min, T feature_max){

    size_t size_min = buf.itemsize * cols;
    size_t size_max = buf.itemsize * cols;

    // max blockdim is 65536,so max col not bigger than 65536*65536*256.
    unsigned int threaddim = 32;
    int blockdim = MAX(ceil(sqrt( ceil(cols/threaddim) )),1);
    dim3 grid_size(blockdim, blockdim);
    dim3 block_size(1,threaddim);


    T *min_d = device_malloc<T>(size_min);
    T *max_d = device_malloc<T>(size_max);

    T min_val = std::numeric_limits<T>::min();
    T max_val = std::numeric_limits<T>::max();

    get_minmax<T><<<grid_size, block_size>>>((T *)buf.ptr_device, min_d, max_d, cols, rows, min_val, max_val);
    hipDeviceSynchronize();
    minmax_scale_cuda<T><<<grid_size, block_size>>>((T *)buf.ptr_device, min_d, max_d,
                                                     cols, rows, feature_min, feature_max);

    device_free(min_d);
    device_free(max_d);
}


int
minmax_scale_cpu(Buf &buf, float feature_min, float feature_max){

    ssize_t ndim = buf.ndim;
    ssize_t rows, cols;
    auto shape = buf.shape;
    auto tmp = buf.rows_cols();
    auto rows = tmp[0];
    auto cols  = tmp[1];

    switch (buf.dtype){
        case Dtype::INT:
            _minmax_scale_cpu<int>(buf, rows, cols, (int)feature_min, (int)feature_max);
            break;
        case Dtype::FLOAT:
            _minmax_scale_cpu<float>(buf, rows, cols, (float)feature_min, (float)feature_max);
            break;

    }
    return 0;
}

//
int
main(){

    float mat[4][2] = {{-1,2},{-0.5,6},{0,10},{1,18}};

    Buf data_buf = Buf( &mat[0][0], 
                   sizeof(float), 
                   Dtype::FLOAT, 
                   2, 
                   {4,2},
                   {2,1}
                   );

    // 1 - copy to device
    host_to_device(data_buf);
    // 2 - get two min max vector
    min_max_vec = get_minmax(buf);
    // 3 - get minmax_scale for train


    // 4 - copy test to device
    // 5 - use two min max vector 
    // 6 - get minmax_scale for test

    // 7 - get result of knn
    minmax_scale_cpu(data_buf, 3, 6);
    device_to_host(data_buf);


    //=================
    from ml4gpu import knn
    knn = KNN()
    knn.fit(train,labels)
    knn.predict(test)
    
    for (int i=0; i<4; i++){
        for (int j=0; j<2; j++){
           printf("%d %d vale %f\n",i,j,mat[i][j]);
        }
    }
}
