#include "hip/hip_runtime.h"
/*
the file imitates sklearn.preprocessing.data
*/
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<limits>
#include<assert.h>
#include<cmath>

#include "ML/preprocessing/data.h"

#include "common/helper.h"
#include "common/buffer_info.h"
#include "common/malloc_free.h"

#define MAX(x,y) ((x)>(y) ? (x): (y))
#define MIN(x,y) ((x)>(y) ? (y): (x))


/* get the [n by m] matrix's maxValue vector and minValue vector by col dimension,
means maxVal vector is [1 by m], minVal vector is [1 by m]*/
template<class T> __global__ void
get_minmax(T *mat, T *min, T *max, unsigned int cols, unsigned int rows, T min_val, T max_val){
    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int thread_idx = (gridDim.x * blockDim.x)*idy +idx;

    // launch col numbers cuda threads to handle the matrix.
    // each cuda thread handle one col dimension
    if (thread_idx < cols){
        T min_l = max_val;
        T max_l = min_val;
    
        T tmp = (T)0;
        for (unsigned int i = 0; i < rows; i++){
            tmp = *(mat + cols*i + thread_idx);
            min_l = MIN(tmp, min_l);
            max_l = MAX(tmp, max_l);
        } 
        min[thread_idx] = min_l;
        max[thread_idx] = max_l;
    }
}

/*normaliza the [n by m] matrix, use col cuda threads*/
template<class T> __global__ void
minmax_scale_cuda(T *mat, T *min, T *max, unsigned int cols, unsigned int rows, T feature_min, T feature_max){
/* feature_range should bigger than 0*/
    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int thread_idx = (gridDim.x * blockDim.x)*idy +idx;

    if (thread_idx < cols){
       T min_l = min[thread_idx];
       T max_l = max[thread_idx];

       T range = max_l - min_l;
       
       T feature_range_l = feature_max - feature_min; // should not be 0, if it's 0, then replace with 1
       assert(feature_range_l > 0 );
       float scale = feature_range_l / range; 
       float min_  = feature_min - min_l * scale;
       
       T tmp = (T)0;
       for(unsigned int i = 0; i<rows; i++){
           tmp = *(mat + cols*i + thread_idx);
           tmp *= scale;
           tmp += min_;
           *(mat + cols*i + thread_idx) = (T)tmp;

       }

    }

}

template<class T> void
_minmax_scale_cpu(Buf &buf, unsigned int rows, unsigned int cols, T feature_min, T feature_max){

    size_t size_min = buf.itermsize * cols;
    size_t size_max = buf.itermsize * cols;

    // max blockdim is 65536,so max col not bigger than 65536*65536*256.
    unsigned int threaddim = 32;
    int blockdim = MAX(ceil(sqrt( ceil(cols/threaddim) )),1);
    dim3 grid_size(blockdim, blockdim);
    dim3 block_size(1,threaddim);

    host_to_device(buf);

    T *min_d = device_malloc<T>(size_min);
    T *max_d = device_malloc<T>(size_max);

    T min_val = std::numeric_limits<T>::min();
    T max_val = std::numeric_limits<T>::max();

    get_minmax<T><<<grid_size, block_size>>>(buf.ptr_device, min_d, max_d, cols, rows, min_val, max_val);
    hipDeviceSynchronize();
    minmax_scale_cuda<T><<<grid_size, block_size>>>(buf.ptr_device, min_d, max_d,
                                                     cols, rows, feature_min, feature_max);

    device_to_host(buf);
    device_free(min_d);
    device_free(max_d);
}

int
minmax_scale_cpu(Buf &buf, float feature_min, float feature_max){

 

    ssize_t ndim = buf.ndim;
    ssize_t rows, cols;
    auto shape = buf.shape;
    switch (ndim){
        case 0:
            assert("current version donot support scalar value!" == 0);
            break;
        case 1:
            rows = shape.size() == 2 ? shape[0] : 0;
            cols = shape.size() == 2 ? shape[1] : shape[0];
            break;
        default:
            rows = shape[0];
            cols = shape[1];
            break;
    }


    switch (buf.type){
        case "int":
            _minmax_scale_cpu<int>(buf, rows, cols, (int)feature_min, (int)feature_max);
            break;
        case "float":
            _minmax_scale_cpu<float>(buf, rows, cols, (float)feature_min, (float)feature_max);
            break;

    }

    return 0;
}

//
//int
//main(){
//
//    float mat[4][2] = {{-1,2},{-0.5,6},{0,10},{1,18}};
//
//    float *p = &mat[0][0];
//    minmax_scale_cpu<float>(p, 2, 4, 3, 6);
//
//    for (int i=0; i<4; i++){
//        for (int j=0; j<2; j++){
//           printf("%d %d vale %f\n",i,j,mat[i][j]);
//        }
//    }
//}
