#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <limits>
#include <hip/hip_runtime.h>
#include "common/malloc_free.h"
#include "ML/knn/sort.h"

typedef unsigned int u32;
#define MAX_NUM_LISTS 128


// radix sort only support unsigned int.
// when handling float, we can `unsigned int after = (unsigned int)(float before*1000)`

template<typename T> __device__ void
radix_sort2(T  * const sort_tmp,
            u32  * sort_ind,
            const u32  num_lists,
            const u32  num_elements,
            const u32  tid,
            T  * const sort_tmp_1,
            u32  *sort_ind_1,
            u32 precision = 1,
            u32 bit_size = 32){

    // num_lists must be even
    assert(num_lists % 2 == 0);

    // init the ind vector
    u32 i_tid = 0+tid;
    for(u32  i = 0;i+tid < num_elements; i+= num_lists){
        i_tid = i+tid;
        sort_ind[i_tid] = i_tid;
    }
    

    for (u32  bit = 0; bit < bit_size; bit++){

        const u32  bit_mask = (1 << bit);
        u32  base_cnt_0 = 0;
        u32  base_cnt_1 = 0;


        i_tid = 0+tid;
        for(u32  i = 0;  i+tid < num_elements; i+= num_lists){

          i_tid = i+tid;
          // get the val, then if float, we should preserve precision,e.g, 1 10 100 1000
          const T elem_tmp = sort_tmp[i_tid];
          // radix sort only support unsigned int
          const u32 elem = (u32)(elem_tmp*precision);
//          const u32  elem = sort_tmp[i_tid];

//          if(tid == 0 && bit == 1)
//              printf(" [%f %d] ",elem_tmp, elem );
          const u32  ind = sort_ind[i_tid];

          if ((elem & bit_mask) > 0){
              sort_tmp_1[base_cnt_1+tid] = elem_tmp;
              // handle the index
              sort_ind_1[base_cnt_1+tid] = ind;
              base_cnt_1 += num_lists;
          }else{
              sort_tmp[base_cnt_0+tid] = elem_tmp;
              // handle the index
              sort_ind[base_cnt_0+tid] = ind;
              base_cnt_0 += num_lists;
          }
        }

        // copy data back to source from the one's list 
        /*cannot use sort_ind replace sort_tmp_1, because after some iter,
        the value of the ind has not been the origin value.
        */
        i_tid = 0+tid;
        for(u32  i = 0; i<base_cnt_1; i += num_lists){
            i_tid = i+tid;
            sort_tmp[base_cnt_0+i_tid] = sort_tmp_1[i_tid];
            sort_ind[base_cnt_0+i_tid] = sort_ind_1[i_tid];
        }
    }

}


template<typename T> __device__ void
merge_array(const T * const src_array,
            const u32 * const src_ind_array,
            T * const dest_array,
            u32 * const dest_ind_array,
            const u32 num_lists,
            const u32 num_elements,
            const u32 tid,
            const T max_val ){

    // num_lists must be even
    assert(num_lists % 2 == 0);
    //const u32 num_elements_per_list = num_elements / num_lists;
  
    __shared__ u32 list_indexes[MAX_NUM_LISTS];
    __shared__ T reduction_val[MAX_NUM_LISTS];
    __shared__ u32 reduction_idx[MAX_NUM_LISTS];

    // 1 - clear the working set
    list_indexes[tid] = 0; // current tid had handled elems
    reduction_val[tid] = 0;
    reduction_idx[tid] = 0;
    __syncthreads();

    for(u32 i = 0; i < num_elements; i++){

       u32 tid_max = num_lists >> 1;   
       T data;

       // 2 - for current thread, get data
       // whether current tid has handle the num of elems
       //if (list_indexes[tid] < num_elements_per_list){
       if (tid+list_indexes[tid]*num_lists < num_elements){
           // cur data index in src array
           const u32 src_idx = tid + (list_indexes[tid] * num_lists);
           data = src_array[src_idx];
       }else{
           data = max_val; // data = 0xFFFFFFFF;
       }

       //store the current data value and index
       reduction_val[tid] = data;
       reduction_idx[tid] = tid;

       // wait for all threads to copy
       __syncthreads();

       
       // 3 - reduce from num_lists to one thread zero
       while(tid_max != 0){
           // gradually reduce tid_max from num_lists to zero

           if(tid < tid_max){
               // calculate the index of  the other half
               // the id of other thread
               const u32 val2_idx = tid + tid_max;
               // read in the other half
               const T val2 = reduction_val[val2_idx];

               //if this half is bigger
               if (reduction_val[tid] > val2){
                   // the store the smaller value
                   reduction_val[tid] = val2;
                   reduction_idx[tid] = reduction_idx[val2_idx];
               }

           }

           // divide tid_max by two
           tid_max >>= 1;

           __syncthreads();
       }

       // 4 - only 0 ind can store dest value
       if (tid == 0){

           // store the winning value
           dest_array[i] = reduction_val[0];

           const u32 ind_idx = list_indexes[reduction_idx[0]]*num_lists + reduction_idx[0];
           dest_ind_array[i] = src_ind_array[ind_idx];

           // increment the list pointer for this thread
           list_indexes[reduction_idx[0]] ++ ;
       }
       
       // wait for tid zero
       __syncthreads();

    }

}


template<typename T> __global__ void
sort_by_rows(T  *mat, u32  *ind_mat, size_t rows, size_t cols, 
             T  * tmp_1, u32  *ind_1, u32 num_lists, u32 precision, T max_val){

    //num_lists should be 256;
    u32 bx = blockIdx.x;
    u32 tx = threadIdx.x;

    radix_sort2<T>(mat+bx*cols, ind_mat+bx*cols,
              num_lists,cols,tx,
              tmp_1+bx*cols, 
              ind_1+bx*cols ,
              precision);
        
    __syncthreads();


    merge_array<T>(mat+bx*cols,ind_mat+bx*cols,
                tmp_1+bx*cols, ind_1+bx*cols,
                num_lists,cols,tx, max_val);
}


template<typename T> void
sort_by_rows_cpu(T  *mat, u32  *ind_mat, size_t rows, size_t cols, u32 precision, T max_val){
    
    size_t size = sizeof(T)*rows*cols;
    size_t size1 = sizeof(u32)*rows*cols;

    // 2function
    T *mat_d = host_to_device_malloc(mat, size);
    u32 *ind_mat_d = host_to_device_malloc(ind_mat, size1);

    // result of two buffer
    T *tmp_1 = device_malloc<T>(size);
    u32 *ind_1 = device_malloc<u32>(size1);
     
    u32 num_lists = MAX_NUM_LISTS;
    dim3 grid(rows);
    dim3 block(num_lists);
    sort_by_rows<T><<<grid,block>>>(mat_d, ind_mat_d, rows, cols,tmp_1,ind_1,num_lists, precision, max_val);


    //2 function
    device_free<T>(mat_d);
    device_free<u32>(ind_mat_d);

    device_to_host_free(mat, tmp_1, size);
    device_to_host_free(ind_mat, ind_1, size1);

//    printf("======================\n");
//    for(int i = 0; i<cols;i++){
//       printf(" [%f %d] ",mat[i],ind_mat[i]);
//    }
} 

template<typename T> void
sort_by_rows_gpu(T  *mat_d, u32  *ind_mat_d, size_t rows, size_t cols, u32 precision, T max_val){

    size_t size = sizeof(T)*rows*cols;
    size_t size1 = sizeof(u32)*rows*cols;


    // result of two buffer
    T *tmp_1 = device_malloc<T>(size);
    u32 *ind_1 = device_malloc<u32>(size1);

    u32 num_lists = MAX_NUM_LISTS;
    dim3 grid(rows);
    dim3 block(num_lists);
    sort_by_rows<T><<<grid,block>>>(mat_d, ind_mat_d, rows, cols,tmp_1,ind_1,num_lists, precision, max_val);


    device_to_device(mat_d, tmp_1, size);
    device_to_device(ind_mat_d, ind_1, size1);

    device_free<T>(tmp_1);
    device_free<u32>(ind_1);

}

void
sort_by_rows(float *mat, u32 *ind_mat, size_t rows, size_t cols, u32 precision){

    float max_val = std::numeric_limits<float>::max();
    sort_by_rows_gpu<float>(mat, ind_mat, rows, cols, precision, max_val);
}


void
sort_by_rows(u32 *mat, u32 *ind_mat, size_t rows, size_t cols, u32 precision){
    u32 max_val = std::numeric_limits<u32>::max();
    sort_by_rows_gpu<u32>(mat, ind_mat, rows, cols, precision, max_val);
    
}


//int
//main(){
//
//    size_t cols = 200;
//    size_t  rows = 1;
//    size_t size = sizeof(u32 )*cols*rows;
//    float  *mat = (float  *)malloc(size);
//
//    u32  *ind_mat = (u32  *)malloc(size);
//
//    for(int i=0; i<cols; i++){
//        mat[i] = cols-i;
//       // mat[i+cols] = cols-i;
//    }
//    float *mat_d = host_to_device_malloc(mat,size);
//    u32 *ind_mat_d = host_to_device_malloc(ind_mat,size);
//
//    u32 precision = 1;
//    float max_val = std::numeric_limits<float>::max();
//    
//    sort_by_rows_gpu<float>(mat_d, ind_mat_d, rows, cols, precision,max_val);
//
//
//    device_to_host_free(mat,mat_d,size);
//    device_to_host_free(ind_mat,ind_mat_d,size);
//    for(int i=0; i<cols; i++){
//        printf("mat %f\n",mat[i]);
//    }
//    free(mat);
//    free(ind_mat);
//
//}
