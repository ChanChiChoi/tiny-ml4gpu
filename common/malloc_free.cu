#include <typeinfo>
#include <assert.h>
#include <string>
#include "common/malloc_free.h"
#include "common/helper.h"
#include "common/common.h"
#include "common/buffer_info_ex.h"



template<class T> T *
device_malloc(size_t size){
    
    T *pdevice = NULL;
    CHECK_CALL(hipMalloc((void **)&pdevice, size));
    return pdevice;
}


void
host_to_device(Buf &buf){

    size_t size = buf.itemsize * buf.size;

    // call device_malloc for malloc buffer on device;
    switch (buf.format){
        case std::string(1,'f'):
            buf.ptr_device = device_malloc<float>(size);
            break;
        default:
            throw std::runtime_error("current version only support float32!");
            break;
    }

    // copy host data to device;

    CHECK_CALL(hipMemcpy(buf.ptr_device, buf.ptr, size, hipMemcpyHostToDevice));
}
 
/* */

template <class T> void
device_free(T *pdevice){

    CHECK_CALL(hipFree(pdevice));
}

void
device_to_host(Buf &buf){
    size_t size = buf.itemsize * buf.size;

    CHECK_CALL(hipMemcpy(buf.ptr_host, buf.ptr_device, size, hipMemcpyDeviceToHost));

    switch (buf.dtype){
        case Dtype::INT:
            device_free<int>((int *)buf.ptr_device);
            break;
        case Dtype::FLOAT:
            device_free<float>((float *)buf.ptr_device);
            break;
        default:
            assert("current version not support other types, except int and float!" == 0);
            break;
    }

    buf.ptr_device = NULL;
}
