#include <typeinfo>
#include <assert.h>
#include "common/malloc_free.h"
#include "common/helper.h"
#include "common/common.h"
#include "common/buffer_info.h"



template<class T> T *
device_malloc(size_t size){
    
    T *pdevice = NULL;
    CHECK_CALL(hipMalloc((void **)&pdevice, size));
    return pdevice;
}


void 
host_to_device(Buf &buf){
    
    size_t size = buf.itemsize * buf.size; 

    // call device_malloc for malloc buffer on device;
    switch (buf.dtype){
        case Dtype::INT:
            buf.ptr_device = device_malloc<int>(size);
            break;

        case Dtype::FLOAT:
            buf.ptr_device = device_malloc<float>(size);
            break;

        default:
            assert("current version not support other types, except int and float!" == 0);
            break;
    }

    // copy host data to device;

    CHECK_CALL(hipMemcpy(buf.ptr_device, buf.ptr_host, size, hipMemcpyHostToDevice));
   
}
 
/* */

template <class T> void
device_free(T *pdevice){

    CHECK_CALL(hipFree(pdevice));
}

void
device_to_host(Buf &buf){
    size_t size = buf.itemsize * buf.size;

    CHECK_CALL(hipMemcpy(buf.ptr_host, buf.ptr_device, size, hipMemcpyDeviceToHost));

    switch (buf.dtype){
        case Dtype::INT:
            device_free<int>((int *)buf.ptr_device);
            break;
        case Dtype::FLOAT:
            device_free<float>((float *)buf.ptr_device);
            break;
        default:
            assert("current version not support other types, except int and float!" == 0);
            break;
    }

    buf.ptr_device = NULL;
}
